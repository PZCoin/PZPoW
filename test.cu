
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <string>

#define MEM_SZ (1LL<<24)
#define NUM_THREADS 1

__device__ void rng(long long *seed, long long &res) {
	*seed = (*seed * 0x5DEECE66DLL + 0xBLL) & ((1LL << 48) - 1);
	res = *seed;
}

__global__ void fill_memory(unsigned char *memory, long long *seed) {
	for (unsigned int i = 0; i < (MEM_SZ>>3); i++) {
		long long res;
		rng(seed, res);
		((uint64_t *)memory)[i] = res;
	}
}

__global__ void gen_nonce(unsigned int idx, unsigned char *str, unsigned char *memory, bool print=false) {
	for (int i = 0; i < 32; i += 2) {
		str[i] = "0123456789abcdef"[memory[idx] >> 4];
		str[i+1] = "0123456789abcdef"[memory[(idx += 997525853) %= MEM_SZ] & 0xf];
	}
	if (print) 
		printf("%s\n", str);
}

__global__ void zero(unsigned char *str) {
	str[32] = 0;
}

__global__ void ihatemylife(long long tmp, long long *val) {
	*val = tmp;
}

int main() {
	unsigned char *memory;
	auto res = hipMalloc(&memory, MEM_SZ);
	if (res != hipSuccess) {
		std::cout << "Error allocating memory: " << hipGetErrorString(res) << "\n";
		return 1;
	}
	std::cout << "Seed: ";
	long long *seed, tmp;
	res = hipMalloc(&seed, 8);
	if (res != hipSuccess) {
		std::cout << "Error allocating memory: " << hipGetErrorString(res) << "\n";
		return 1;
	}
	std::cin >> tmp;
	ihatemylife<<<1, 1>>>(tmp, seed);
	std::cout << "Filling memory...\n";
	clock_t start = clock();
	fill_memory<<<1, 1>>>(memory, seed);
	// wait for kernel to finish
	hipDeviceSynchronize();
	clock_t end = clock();
	std::cout << "Success! " << ((double)(end-start) / CLOCKS_PER_SEC) << " seconds elapsed.\n";
	std::cout << "Testing nonce generation speed...\n";
	unsigned char *tmpnonce;
	res = hipMalloc(&tmpnonce, 33);
	if (res != hipSuccess) {
		std::cout << "Error allocating memory: " << hipGetErrorString(res) << "\n";
		return 1;
	}
	zero<<<1, 1>>>(tmpnonce);
	// error is below here
	start = clock();
	for (int i = 0; i < 33554432; i++) {
		gen_nonce<<<1, NUM_THREADS>>>(i, tmpnonce, memory);
	}
	hipDeviceSynchronize();
	end = clock();
	// errir is above here
	res = hipGetLastError();
	if (res != hipSuccess) {
		std::cout << "Error: " << hipGetErrorString(res) << "\n";
		return 1;
	}
	double speed = (double)CLOCKS_PER_SEC / (end - start);
	speed *= NUM_THREADS;
	std::cout << "Speed: " << std::fixed << speed << " GB / s\n";
	std::cout << "Speed: " << speed * 33.554432 << " MNonces / sec\n";
	for (int i = 0; i < 4; i++) {
		gen_nonce<<<1, 1>>>(i, tmpnonce, memory, true);
		hipDeviceSynchronize();
	}
	// get last error
	res = hipGetLastError();
	if (res != hipSuccess) {
		std::cout << "Error: " << hipGetErrorString(res) << "\n";
		return 1;
	}
	return 0;
}