
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <string>

#define MEM_SZ (1LL<<32)
#define NUM_THREADS 1024

__device__ void rng(long long *seed, long long &res) {
	*seed = (*seed * 0x5DEECE66DLL + 0xBLL) & ((1LL << 48) - 1);
	res = *seed;
}

__global__ void fill_memory(unsigned char *memory, long long *seed) {
	for (unsigned int i = 0; i < (MEM_SZ>>3); i++) {
		long long res;
		rng(seed, res);
		((uint64_t *)memory)[i] = res;
	}
}

__global__ void gen_nonce(unsigned int idx, unsigned char *str, unsigned char *memory) {
	#if MEM_SZ != (1LL<<32)
	idx %= MEM_SZ; // this is fucking dumb i wasted 4 hours debugging this shit
	#endif
	for (int i = 0; i < 32; i += 2) {
		str[i] = "0123456789abcdef"[memory[idx] >> 4];
		str[i+1] = "0123456789abcdef"[memory[(idx += 997525853) 
		#if MEM_SZ != (1LL<<32)
		%= MEM_SZ
		#endif
		] & 0xf];
	}
	long long res;
	long long val = (long long)(idx^0xff2871<<2)+0xf3;
	long long *seed = &val;
	rng(seed, res);
	((uint64_t *)str)[4] = res; // [4] is [32]
	for (int i = 32; i < 40; i += 2) {
		str[i] = "0123456789abcdef"[str[i] >> 4];
		str[i+1] = "0123456789abcdef"[str[i] & 0xf];
	}
}

__global__ void zero(unsigned char *str) {
	str[40] = 0;
}

__global__ void ihatemylife(long long tmp, long long *val) {
	*val = tmp;
}

int main() {
	unsigned char *memory;
	auto res = hipMalloc(&memory, MEM_SZ);
	if (res != hipSuccess) {
		std::cout << "Error allocating memory: " << hipGetErrorString(res) << "\n";
		return 1;
	}
	std::cout << "Seed: ";
	long long *seed, tmp;
	res = hipMalloc(&seed, 8);
	if (res != hipSuccess) {
		std::cout << "Error allocating memory: " << hipGetErrorString(res) << "\n";
		return 1;
	}
	std::cin >> tmp;
	ihatemylife<<<1, 1>>>(tmp, seed);
	std::cout << "Filling memory...\n";
	clock_t start = clock();
	fill_memory<<<1, 1>>>(memory, seed);
	// wait for kernel to finish
	hipDeviceSynchronize();
	clock_t end = clock();
	std::cout << "Success! " << ((double)(end-start) / CLOCKS_PER_SEC) << " seconds elapsed.\n";
	std::cout << "Testing nonce generation speed...\n";
	unsigned char *tmpnonce;
	res = hipMalloc(&tmpnonce, 41);
	if (res != hipSuccess) {
		std::cout << "Error allocating memory: " << hipGetErrorString(res) << "\n";
		return 1;
	}
	zero<<<1, 1>>>(tmpnonce);
	start = clock();
	for (int i = 0; i < 26843546; i++) {
		gen_nonce<<<1, NUM_THREADS>>>(i, tmpnonce, memory);
		hipDeviceSynchronize();
	}
	end = clock();
	res = hipGetLastError();
	if (res != hipSuccess) {
		std::cout << "Error: " << hipGetErrorString(res) << "\n";
		return 1;
	}
	double speed = (double)CLOCKS_PER_SEC / (end - start);
	speed *= NUM_THREADS;
	std::cout << "Success! Took " << ((double)(end-start) / CLOCKS_PER_SEC) << " seconds.\n";
	std::cout << "Speed: " << std::fixed << speed << " GB / s\n";
	std::cout << "Speed: " << speed * 26.8435456 << " MNonces / sec\n";
	char *tmpnonce2 = new char[41];
	for (int i = 0; i < 4; i++) {
		gen_nonce<<<1, 1>>>(i, tmpnonce, memory);
		hipDeviceSynchronize();
		hipMemcpy(tmpnonce2, tmpnonce, 41, hipMemcpyDeviceToHost);
		std::cout << tmpnonce2 << '\n';
	}
	// get last error
	res = hipGetLastError();
	if (res != hipSuccess) {
		std::cout << "Error: " << hipGetErrorString(res) << "\n";
		return 1;
	}
	return 0;
}